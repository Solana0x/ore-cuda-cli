#include <stdlib.h>
#include <string.h>
#include <hashx.h>
#include "context.h"
#include "compiler.h"
#include "program.h"

#define STRINGIZE_INNER(x) #x
#define STRINGIZE(x) STRINGIZE_INNER(x)

#ifndef HASHX_SALT
#define HASHX_SALT "HashX v1"
#endif

// Device constant for Blake2b parameters
__device__ const blake2b_param hashx_blake2_params = {
    64, 0, 1, 1, 0, 0, 0, 0, { 0 }, STRINGIZE(HASHX_SALT), { 0 }
};

hashx_ctx* hashx_alloc(hashx_type type) {
    hashx_ctx* ctx = NULL;

    // Allocate unified memory for context and check for errors
    hipError_t err = hipMallocManaged(&ctx, sizeof(hashx_ctx));
    if (err != hipSuccess || ctx == NULL) {
        fprintf(stderr, "Failed to allocate memory for hashx_ctx: %s\n", hipGetErrorString(err));
        return NULL;
    }

    // Initialize pointers to NULL to avoid undefined behavior
    ctx->code = NULL;
    ctx->program = NULL;

    // Allocate memory based on the type of context
    if (type & HASHX_COMPILED) {
        if (!hashx_compiler_init(ctx)) {
            hipFree(ctx);
            return NULL;
        }
        ctx->type = HASHX_COMPILED;
    } else {
        err = hipMallocManaged(&ctx->program, sizeof(hashx_program));
        if (err != hipSuccess || ctx->program == NULL) {
            fprintf(stderr, "Failed to allocate memory for hashx_program: %s\n", hipGetErrorString(err));
            hipFree(ctx);
            return NULL;
        }
        ctx->type = HASHX_INTERPRETED;
    }

#ifdef HASHX_BLOCK_MODE
    // Direct initialization of blake2b_param structure (faster than memcpy)
    ctx->params = hashx_blake2_params;
#endif

    return ctx;
}

void hashx_free(hashx_ctx* ctx) {
    if (ctx != NULL && ctx != HASHX_NOTSUPP) {
        if (ctx->type & HASHX_COMPILED) {
            hashx_compiler_destroy(ctx);
        } else {
            hipFree(ctx->program);
        }
        hipFree(ctx);
    }
}
