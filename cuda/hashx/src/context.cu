#include <stdlib.h>
#include <string.h>
#include <../include/hashx.h>
#include "context.h"
#include "compiler.h"
#include "program.h"

#define STRINGIZE_INNER(x) #x
#define STRINGIZE(x) STRINGIZE_INNER(x)

#ifndef HASHX_SALT
#define HASHX_SALT "HashX v1"
#endif

__device__ const blake2b_param hashx_blake2_params = {
    64, 0, 1, 1, 0, 0, 0, 0, { 0 }, STRINGIZE(HASHX_SALT), { 0 }
};

hashx_ctx* hashx_alloc(hashx_type type) {
    hashx_ctx* ctx;

    // Allocate memory for context
    if (hipMallocManaged(&ctx, sizeof(hashx_ctx)) != hipSuccess) {
        return NULL;
    }

    // Initialize the context
    ctx->code = NULL;
    ctx->program = NULL;
    
    // Set the type
    ctx->type = HASHX_UNDEFINED;  // Replace 'HASHX_UNDEFINED' with the appropriate value

    // Choose the appropriate type
    if (type & HASHX_COMPILED) {
        if (!hashx_compiler_init(ctx)) {
            hipFree(ctx);
            return NULL;
        }
        ctx->type = HASHX_COMPILED;
    } else {
        if (hipMallocManaged(&ctx->program, sizeof(hashx_program)) != hipSuccess) {
            hipFree(ctx);
            return NULL;
        }
        ctx->type = HASHX_INTERPRETED;
    }

#ifdef HASHX_BLOCK_MODE
    hipMemcpy(&ctx->params, &hashx_blake2_params, sizeof(blake2b_param), hipMemcpyDefault);
#endif

    return ctx;
}

void hashx_free(hashx_ctx* ctx) {
    if (ctx != NULL && ctx != HASHX_NOTSUPP) {
        if (ctx->type & HASHX_COMPILED) {
            hashx_compiler_destroy(ctx);
        } else if (ctx->program != NULL) {
            hipFree(ctx->program);
        }
        hipFree(ctx);
    }
}
