#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <vector>
#include "drillx.h"
#include "equix/include/equix.h"
#include "hashx/include/hashx.h"
#include "equix/src/context.h"
#include "equix/src/solver.h"
#include "equix/src/solver_heap.h"
#include "hashx/src/context.h"

const int BATCH_SIZE = 8192;
__constant__ int NUM_HASHING_ROUNDS = 1;  // Use __constant__ memory for NUM_HASHING_ROUNDS

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = (call); \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(err); \
        } \
    } while (0)

extern "C" void set_num_hashing_rounds(int rounds) {
    // Updated to set the constant variable in device code
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(NUM_HASHING_ROUNDS), &rounds, sizeof(int)));
}

extern "C" void hash(uint8_t *challenge, uint8_t *nonce, uint64_t *out) {
    MemoryPool memPool(BATCH_SIZE);

    std::vector<uint8_t> seed(40);
    memcpy(seed.data(), challenge, 32);

    for (int i = 0; i < BATCH_SIZE; i++) {
        uint64_t nonce_offset = *((uint64_t*)nonce) + i;
        memcpy(seed.data() + 32, &nonce_offset, 8);
        memPool.ctxs[i] = hashx_alloc(HASHX_INTERPRETED);
        if (!memPool.ctxs[i] || !hashx_make(memPool.ctxs[i], seed.data(), 40)) {
            return;
        }
    }

    int threadsPerBlock = 256;  // Reduced number of threads per block to reduce memory footprint
    int blocksPerGrid = (BATCH_SIZE * INDEX_SPACE + threadsPerBlock - 1) / threadsPerBlock;

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    do_hash_stage0i<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(memPool.ctxs, memPool.hash_space, NUM_HASHING_ROUNDS);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipStreamSynchronize(stream));

    for (int i = 0; i < BATCH_SIZE; i++) {
        CUDA_CHECK(hipMemcpyAsync(out + i * INDEX_SPACE, memPool.hash_space[i], INDEX_SPACE * sizeof(uint64_t), hipMemcpyDeviceToHost, stream));
    }

    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
}

__global__ void do_hash_stage0i(hashx_ctx** ctxs, uint64_t** hash_space, int num_hashing_rounds) {
    uint32_t item = blockIdx.x * blockDim.x + threadIdx.x;
    if (item < BATCH_SIZE * INDEX_SPACE) {
        uint32_t batch_idx = item / INDEX_SPACE;
        uint32_t i = item % INDEX_SPACE;

        for (int round = 0; round < num_hashing_rounds; ++round) {
            hash_stage0i(ctxs[batch_idx], hash_space[batch_idx], i);
        }
    }
}

extern "C" void solve_all_stages(uint64_t *hashes, uint8_t *out, uint32_t *sols, int num_sets) {
    uint64_t *d_hashes;
    solver_heap *d_heaps;
    equix_solution *d_solutions;
    uint32_t *d_num_sols;

    // Use hipMallocManaged for memory that can be shared between host and device
    CUDA_CHECK(hipMallocManaged(&d_hashes, num_sets * INDEX_SPACE * sizeof(uint64_t)));
    CUDA_CHECK(hipMallocManaged(&d_heaps, num_sets * sizeof(solver_heap)));
    CUDA_CHECK(hipMallocManaged(&d_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution)));
    CUDA_CHECK(hipMallocManaged(&d_num_sols, num_sets * sizeof(uint32_t)));

    CUDA_CHECK(hipMemcpy(d_hashes, hashes, num_sets * INDEX_SPACE * sizeof(uint64_t), hipMemcpyHostToDevice));

    int threadsPerBlock = 256;  // Adjusted to a more optimal value
    int blocksPerGrid = (num_sets + threadsPerBlock - 1) / threadsPerBlock;

    solve_all_stages_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_hashes, d_heaps, d_solutions, d_num_sols);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipDeviceSynchronize());

    // Use unified memory to avoid multiple memcpy calls
    for (int i = 0; i < num_sets; i++) {
        sols[i] = d_num_sols[i];
        if (d_num_sols[i] > 0) {
            memcpy(out + i * sizeof(equix_solution), &d_solutions[i * EQUIX_MAX_SOLS], sizeof(equix_solution));
        }
    }

    CUDA_CHECK(hipFree(d_hashes));
    CUDA_CHECK(hipFree(d_heaps));
    CUDA_CHECK(hipFree(d_solutions));
    CUDA_CHECK(hipFree(d_num_sols));
}
