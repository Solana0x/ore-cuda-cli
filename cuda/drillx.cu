#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "drillx.h"
#include "equix/include/equix.h"
#include "hashx/include/hashx.h"
#include "equix/src/context.h"
#include "equix/src/solver.h"
#include "equix/src/solver_heap.h"
#include "hashx/src/context.h"

const int BATCH_SIZE = 16384; // Large batch size
const int NUM_HASHING_ROUNDS = 1;

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = (call); \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(err); \
        } \
    } while (0)

extern "C" void set_num_hashing_rounds(int rounds) {
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(NUM_HASHING_ROUNDS), &rounds, sizeof(int)));
}

extern "C" void hash(uint8_t *challenge, uint8_t *nonce, uint64_t *out) {
    // Increase the CUDA heap size to handle larger allocations
    size_t heapSize = 24L * 1024L * 1024L * 1024L; // 24 GB
    CUDA_CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize));

    // Allocate the memory pool on the host
    MemoryPool* memPool = new MemoryPool(BATCH_SIZE);

    uint8_t seed[40];
    memcpy(seed, challenge, 32);

    for (int i = 0; i < BATCH_SIZE; i++) {
        uint64_t nonce_offset = *((uint64_t*)nonce) + i;
        memcpy(seed + 32, &nonce_offset, 8);
        memPool->ctxs[i] = hashx_alloc(HASHX_INTERPRETED);
        if (!memPool->ctxs[i] || !hashx_make(memPool->ctxs[i], seed, 40)) {
            delete memPool; // Clean up before returning
            return;
        }
    }

    int threadsPerBlock = 1024;
    int blocksPerGrid = (BATCH_SIZE * INDEX_SPACE + threadsPerBlock - 1) / threadsPerBlock;

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Perform the hashing on the GPU
    do_hash_stage0i<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(memPool->ctxs, memPool->hash_space, NUM_HASHING_ROUNDS);
    CUDA_CHECK(hipGetLastError());

    // Synchronize the stream to ensure the kernel execution completes
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Ensure hash results are transferred from GPU to CPU
    for (int i = 0; i < BATCH_SIZE; i++) {
        // Copy data from the GPU to the CPU
        CUDA_CHECK(hipMemcpy(out + i * INDEX_SPACE, memPool->hash_space[i], INDEX_SPACE * sizeof(uint64_t), hipMemcpyDeviceToHost));
    }

    // Synchronize after copying the data
    CUDA_CHECK(hipDeviceSynchronize());

    // Destroy the stream
    CUDA_CHECK(hipStreamDestroy(stream));

    // Clean up memory pool
    delete memPool;
}

__global__ void do_hash_stage0i(hashx_ctx** ctxs, uint64_t** hash_space, int num_hashing_rounds) {
    uint32_t item = blockIdx.x * blockDim.x + threadIdx.x;
    if (item < BATCH_SIZE * INDEX_SPACE) {
        uint32_t batch_idx = item / INDEX_SPACE;
        uint32_t i = item % INDEX_SPACE;

        for (int round = 0; round < num_hashing_rounds; ++round) {
            hash_stage0i(ctxs[batch_idx], hash_space[batch_idx], i);
        }
    }
}

extern "C" void solve_all_stages(uint64_t *hashes, uint8_t *out, uint32_t *sols, int num_sets) {
    // Increase the CUDA heap size to handle larger allocations
    size_t heapSize = 24L * 1024L * 1024L * 1024L; // 24 GB
    CUDA_CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize));

    // Use unified memory for automatic paging between CPU and GPU
    uint64_t *d_hashes;
    solver_heap *d_heaps;
    equix_solution *d_solutions;
    uint32_t *d_num_sols;

    CUDA_CHECK(hipMallocManaged(&d_hashes, num_sets * INDEX_SPACE * sizeof(uint64_t)));
    CUDA_CHECK(hipMallocManaged(&d_heaps, num_sets * sizeof(solver_heap)));
    CUDA_CHECK(hipMallocManaged(&d_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution)));
    CUDA_CHECK(hipMallocManaged(&d_num_sols, num_sets * sizeof(uint32_t)));

    CUDA_CHECK(hipMemcpy(d_hashes, hashes, num_sets * INDEX_SPACE * sizeof(uint64_t), hipMemcpyHostToDevice));

    int threadsPerBlock = 1024;
    int blocksPerGrid = (num_sets + threadsPerBlock - 1) / threadsPerBlock;

    solve_all_stages_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_hashes, d_heaps, d_solutions, d_num_sols);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(out, d_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(sols, d_num_sols, num_sets * sizeof(uint32_t), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_hashes));
    CUDA_CHECK(hipFree(d_heaps));
    CUDA_CHECK(hipFree(d_solutions));
    CUDA_CHECK(hipFree(d_num_sols));
}
